#include "hip/hip_runtime.h"
#include "shared.h"

#include <cstdio>
#include <cmath>
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
//#include "hipblas.h"

extern "C"{

/*Reaproveite este kernel pois ele pode ser usado para resolver a parte
 * de deslocamentos de pontos internos de Interec.for*/
__global__ void ghmatecd_kernel(
                           int cone[],
                           FREAL cx[],
                           FREAL cy[],
                           FREAL cz[],
                           FREAL cxm[],
                           FREAL cym[],
                           FREAL czm[],
                           thrust::complex<FREAL> zh[],
                           thrust::complex<FREAL> zg[],
                           FREAL rn[][3],
                           thrust::complex<FREAL> zge,
                           thrust::complex<FREAL> zcs,
                           thrust::complex<FREAL> zcp,
                           FREAL fr,
                           FREAL gi[],
                           FREAL ome[],
                           FREAL c1,
                           FREAL c2,
                           FREAL c3,
                           FREAL c4,
						   FREAL hestdiag[],
						   FREAL gestdiag[],
                           int n,
                           int nbe,
                           int dim_cone,
						   int column_pad,
                           int fast_singular,
                           int* ret
                           );

void cuda_interec1_(int* n,
                    int* nbe,
                    int* npg,
                    int* l,
                    int* np,
					FREAL cxi[],
					FREAL cyi[],
					FREAL czi[],
                    thrust::complex<FREAL>* zge,
                    thrust::complex<FREAL>* zcs,
                    thrust::complex<FREAL>* zcp,
                    FREAL* c1,
                    FREAL* c2,
                    FREAL* c3,
                    FREAL* c4,
                    FREAL* fr,
                    thrust::complex<FREAL> zdfi[],
					thrust::complex<FREAL> zfi[],
                    thrust::complex<FREAL> zdsol[],
                    int* status
                   )
{
	
	dim3 threadsPerBlock(*npg,*npg);
	dim3 numBlocks(*nbe, *l);
	int shared_mem_size = 2*3*3*(*npg)*(*npg)*sizeof(thrust::complex<FREAL>);
	hipError_t error;
    
	thrust::complex<FREAL> zhelem[3][3];
	thrust::complex<FREAL> zgelem[3][3];

	thrust::complex<FREAL>* device_zh;
	thrust::complex<FREAL>* device_zg;
    thrust::complex<FREAL>* device_zdfi;
    thrust::complex<FREAL>* device_zfi;
    thrust::complex<FREAL>* device_zdsol;

	FREAL* device_cxi;
	FREAL* device_cyi;
	FREAL* device_czi;

	int* device_return_status;
	int return_status;

    thrust::complex<FREAL> one(1., 0.);
    thrust::complex<FREAL> zero(0., 0.);
    thrust::complex<FREAL> minus_one(-1., 0.);

    hipblasHandle_t handle;
	hipblasStatus_t stats;

	error = hipMalloc(&device_return_status, sizeof(int));
	cuda_assert(error);

	error = hipMalloc(&device_zh, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	error = hipMalloc(&device_zg, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	error = hipMemset(device_return_status, 0, sizeof(int));
	cuda_assert(error);

	error = hipMemset(device_zh, 0, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	error = hipMemset(device_zg, 0, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	error = hipMalloc(&device_cxi, (*l)*sizeof(FREAL));
	cuda_assert(error);
	
	error = hipMalloc(&device_cyi, (*l)*sizeof(FREAL));
	cuda_assert(error);
	
	error = hipMalloc(&device_czi, (*l)*sizeof(FREAL));
	cuda_assert(error);

	error = hipMemcpy(device_cxi, cxi, (*l)*sizeof(FREAL), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_cyi, cyi, (*l)*sizeof(FREAL), hipMemcpyHostToDevice);
	cuda_assert(error);
	
	error = hipMemcpy(device_czi, czi, (*l)*sizeof(FREAL), hipMemcpyHostToDevice);
	cuda_assert(error);

	ghmatecd_kernel<<<numBlocks, threadsPerBlock, shared_mem_size>>>(
						device_cone,
						device_cx,
						device_cy,
						device_cz,
						device_cxi,
						device_cyi,
						device_czi,
                        device_zh,
                        device_zg,
						(FREAL (*)[3]) device_etas,
						*zge,
						*zcs,
						*zcp,
						*fr,
						device_gi,
						device_ome,
						*c1,
						*c2,
						*c3,
						*c4,
						NULL,
						NULL,
						*nbe,
						*l,
                        *n,
						0,
						0,
						device_return_status
						);
    
	hipDeviceSynchronize();

	error = hipMalloc(&device_zdsol, 3*(*l)*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

    error = hipMalloc(&device_zdfi, 3*(*nbe)*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);
	
	error = hipMalloc(&device_zfi, 3*(*nbe)*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	error = hipMemcpy(device_zdfi, zdfi, 3*(*nbe)*sizeof(thrust::complex<FREAL>), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_zfi, zfi, 3*(*nbe)*sizeof(thrust::complex<FREAL>), hipMemcpyHostToDevice);
	cuda_assert(error);
    hipDeviceSynchronize();

	error = hipMemcpy(&return_status, device_return_status, sizeof(int), hipMemcpyDeviceToHost);
	cuda_assert(error);

	if (return_status != 0)
	{
		fputs("Matriz Singular\n", stderr);
	}

    stats = hipblasCreate(&handle);
	cublas_assert(stats);
  
    if (sizeof(FREAL) == 8)
    {
        stats = hipblasZgemv(handle, HIPBLAS_OP_N, 3*(*l), 3*(*nbe), (hipDoubleComplex*) &one, (hipDoubleComplex*) device_zg, 3*(*l), (hipDoubleComplex*) device_zdfi, 1, (hipDoubleComplex*) &zero, (hipDoubleComplex*) device_zdsol, 1);
        cublas_assert(stats);
        hipDeviceSynchronize();
        
        stats = hipblasZgemv(handle, HIPBLAS_OP_N, 3*(*l), 3*(*nbe), (hipDoubleComplex*) &(minus_one), (hipDoubleComplex*) device_zh, 3*(*l), (hipDoubleComplex*) device_zfi, 1, (hipDoubleComplex*) &one, (hipDoubleComplex*) device_zdsol, 1);
        cublas_assert(stats);
        hipDeviceSynchronize(); 
    }
    else
    {
        stats = hipblasCgemv(handle, HIPBLAS_OP_N, 3*(*l), 3*(*nbe), (hipComplex*) &one, (hipComplex*) device_zg, 3*(*l), (hipComplex*) device_zdfi, 1, (hipComplex*) &zero, (hipComplex*) device_zdsol, 1);
        cublas_assert(stats);
        hipDeviceSynchronize();
        
        stats = hipblasCgemv(handle, HIPBLAS_OP_N, 3*(*l), 3*(*nbe), (hipComplex*) &(minus_one), (hipComplex*) device_zh, 3*(*l), (hipComplex*) device_zfi, 1, (hipComplex*) &one, (hipComplex*) device_zdsol, 1);
        cublas_assert(stats);
        hipDeviceSynchronize(); 
    }

	error = hipMemcpy(zdsol, device_zdsol, 3*(*l)*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
	cuda_assert(error);

    error = hipFree(device_zh);
	cuda_assert(error);
	error = hipFree(device_zg);
	*status = return_status;
	error = hipFree(device_return_status);
	cuda_assert(error);

	error = hipFree(device_cxi);
	cuda_assert(error);
	error = hipFree(device_cyi);
	cuda_assert(error);
	error = hipFree(device_czi);
	cuda_assert(error);
	error = hipFree(device_zfi);
	cuda_assert(error);
	error = hipFree(device_zdfi);
	cuda_assert(error);

	error = hipFree(device_zdsol);
	cuda_assert(error);

	hipblasDestroy(handle);
}
}
