#include "hip/hip_runtime.h"
#include "shared.h"

#include <cstdio>
#include <cmath>
#include <thrust/complex.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

extern "C"{

/*Reaproveite este kernel pois ele pode ser usado para resolver a parte
 * de deslocamentos de pontos internos de Interec.for*/
__global__ void ghmatecd_kernel(
                           int cone[],
                           float cx[],
                           float cy[],
                           float cz[],
                           float cxm[],
                           float cym[],
                           float czm[],
                           thrust::complex<float> zh[],
                           thrust::complex<float> zg[],
                           float rn[][3],
                           thrust::complex<float> zge,
                           thrust::complex<float> zcs,
                           thrust::complex<float> zcp,
                           float fr,
                           float gi[],
                           float ome[],
                           float c1,
                           float c2,
                           float c3,
                           float c4,
                           int npg,
                           int n,
                           int nbe,
                           int interec,
                           int* ret
                           );

void cuda_interec1_(int* nbe,
                    int* npg,
                    int* l,
                    int* np,
					float cxi[],
					float cyi[],
					float czi[],
                    thrust::complex<float>* zge,
                    thrust::complex<float>* zcs,
                    thrust::complex<float>* zcp,
                    float* c1,
                    float* c2,
                    float* c3,
                    float* c4,
                    float* fr,
                    thrust::complex<float>* zgp_,
                    thrust::complex<float>* zhp_,
                    thrust::complex<float> zdfi[],
					thrust::complex<float> zfi[],
                    int* status
                   )
{
	dim3 threadsPerBlock(*npg,*npg);
	dim3 numBlocks(*nbe, *l);
	hipError_t error;
    
	thrust::complex<float> zhelem[3][3];
	thrust::complex<float> zgelem[3][3];

	thrust::complex<float>* device_zh;
	thrust::complex<float>* device_zg;
	float* device_cxi;
	float* device_cyi;
	float* device_czi;

	int* device_return_status;
	int return_status;

	thrust::complex<float> (*zgp)[3*(*l)] = (thrust::complex<float> (*)[3*(*l)]) zgp_;
	thrust::complex<float> (*zhp)[3*(*l)] = (thrust::complex<float> (*)[3*(*l)]) zhp_;

	int i, ii;

	error = hipMalloc(&device_return_status, sizeof(int));
	cuda_assert(error);

	error = hipMalloc(&device_zh, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<float>));
	cuda_assert(error);

	error = hipMalloc(&device_zg, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<float>));
	cuda_assert(error);

	error = hipMemset(device_return_status, 0, sizeof(int));
	cuda_assert(error);

	error = hipMemset(device_zh, 0, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<float>));
	cuda_assert(error);

	error = hipMemset(device_zg, 0, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<float>));
	cuda_assert(error);


	error = hipMalloc(&device_cxi, (*l)*sizeof(float));
	cuda_assert(error);
	
	error = hipMalloc(&device_cyi, (*l)*sizeof(float));
	cuda_assert(error);
	
	error = hipMalloc(&device_czi, (*l)*sizeof(float));
	cuda_assert(error);

	error = hipMemcpy(device_cxi, cxi, (*l)*sizeof(float), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_cyi, cyi, (*l)*sizeof(float), hipMemcpyHostToDevice);
	cuda_assert(error);
	
	error = hipMemcpy(device_czi, czi, (*l)*sizeof(float), hipMemcpyHostToDevice);
	cuda_assert(error);


	hipDeviceSynchronize();

	ghmatecd_kernel<<<numBlocks, threadsPerBlock>>>(
						device_cone,
						device_cx,
						device_cy,
						device_cz,
						device_cxi,
						device_cyi,
						device_czi,
						device_zh,
						device_zg,
						(float (*)[3]) device_etas,
						*zge,
						*zcs,
						*zcp,
						*fr,
						device_gi,
						device_ome,
						*c1,
						*c2,
						*c3,
						*c4,
						*npg,
						*nbe,
						*l,
                        1,
						device_return_status
						);
	hipDeviceSynchronize();

	error = hipMemcpy(&return_status, device_return_status, sizeof(int), hipMemcpyDeviceToHost);
	cuda_assert(error);

	if (return_status != 0)
	{
		fputs("Matriz Singular\n", stderr);
	}

	error = hipMemcpy(zhp_, device_zh, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
	cuda_assert(error);
	error = hipMemcpy(zgp_, device_zg, (3*(*nbe))*(3*(*l))*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
	cuda_assert(error);
/*
	for (int j = 0; j < 3*(*nbe); ++j)
	{
		for (int i = 0; i < 3*(*l); ++i)
		{
			printf("%d %d %f %f\n", i, j, zgp[j][i].real(), zgp[j][i].imag());
		}
	}
*/
	error = hipFree(device_zh);
	cuda_assert(error);
	error = hipFree(device_zg);
	*status = return_status;
	error = hipFree(device_return_status);
	cuda_assert(error);

	error = hipFree(device_cxi);
	cuda_assert(error);
	error = hipFree(device_cyi);
	cuda_assert(error);
	error = hipFree(device_czi);
	cuda_assert(error);
}
}
