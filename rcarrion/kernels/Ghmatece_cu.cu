#include "hip/hip_runtime.h"
#include "shared.h"

#include <cstdio>
#include <cmath>
#include <thrust/complex.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

extern "C"{

void rigid_body_(
            int* nbe,
            int* n,
            float hest_[],
            float hestdiag[][3][3]
        );


__global__ void ghmatece_kernel(
                           int cone[],
                           float cx[],
                           float cy[],
                           float cz[],
                           float cxm[],
                           float cym[],
                           float czm[],
                           float hest[],
                           float rn[][3],
                           float fr,
                           float gi[],
                           float ome[],
                           float c3,
                           float c4,
                           int npg,
                           int n,
                           int nbe,
						   int column_pad,
                           int* ret
                           )
{

	const int ig = threadIdx.y;
	const int jg = threadIdx.x;
	const int ii = blockIdx.y;
	const int jj = blockIdx.x + column_pad;

	const int zgelem_pad = 3*3*npg*npg;
	extern __shared__ float s_[];
	
	int i, j;
	
	float p[4][2], f[4];
	float xj[3][2];
	__shared__ float co[3][4];
	__shared__ float rn_cached[4];
	float p1, p2, p12, g1, g2, sp, sm, rp, rm, det;
	float cxg, cyg, czg, cxp, cyp, czp;
	float j1, j2, j3;
	float r1, r2, r3, r, drn, rd[3];
	float hei;

//Manage the shared memory manually since apparently there is no other way
//to allocate two cubes in dynamic allocated shared memory.
//see https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
#define helem(i, j, k) s_[3*npg*npg*(i) + npg*npg*(j) + (k)]

	int iii, jjj;

	for (iii = 0; iii < 3; ++iii)
		for (jjj = 0; jjj < 3; ++jjj)
			helem(iii, jjj, npg*ig + jg) = 0;

    if (ii != jj)
    {

        if (threadIdx.x < 4 && threadIdx.y == 0)
        {
            co[0][threadIdx.x] = cx[cone[n*threadIdx.x + jj] - 1];
            co[1][threadIdx.x] = cy[cone[n*threadIdx.x + jj] - 1];
            co[2][threadIdx.x] = cz[cone[n*threadIdx.x + jj] - 1];
            rn_cached[threadIdx.x] = rn[jj][threadIdx.x];
        }
        __syncthreads();

        cxp = cxm[ii];
        cyp = cym[ii];
        czp = czm[ii];

        g2 = gi[jg];
        p2 = ome[jg];
        sp = 1 + g2;
        sm = 1 - g2;
        p[0][0] = -0.25f*sm;
        p[1][0] =  0.25f*sm;
        p[2][0] =  0.25f*sp;
        p[3][0] = -0.25f*sp;

        g1 = gi[ig];
        p1 = ome[ig];
        rp = 1 + g1;
        rm = 1 - g1;
        f[0] = 0.25f*rm*sm;
        f[1] = 0.25f*rp*sm;
        f[2] = 0.25f*rp*sp;
        f[3] = 0.25f*rm*sp;
        p[0][1] = -0.25f*rm;
        p[1][1] = -0.25f*rp;
        p[2][1] = 0.25f*rp;
        p[3][1] = 0.25f*rm;

        
       
        for (iii = 0; iii < 2; ++iii)
        {
            for (jjj = 0; jjj < 3; ++jjj)
            {
                xj[jjj][iii] = p[0][iii]*co[jjj][0] + p[1][iii]*co[jjj][1]+ p[2][iii]*co[jjj][2] + p[3][iii]*co[jjj][3];
            }
        }
        

        j1 = xj[1][0]*xj[2][1]-xj[1][1]*xj[2][0];
        j2 = xj[0][1]*xj[2][0]-xj[0][0]*xj[2][1];
        j3 = xj[0][0]*xj[1][1]-xj[0][1]*xj[1][0];

        det = sqrt(j1*j1 + j2*j2 + j3*j3);

        if (det < 1e-5)
        {
            *ret = 1;
            return;
        }


        cxg = 0;
        cyg = 0;
        czg = 0;

        for (iii = 0; iii < 4; ++iii)
        {
            cxg = cxg + co[0][iii]*f[iii];
            cyg = cyg + co[1][iii]*f[iii];
            czg = czg + co[2][iii]*f[iii];
        }

        r1    = cxg - cxp;
        r2    = cyg - cyp;
        r3    = czg - czp;

        r     = sqrt(r1*r1 + r2*r2 + r3*r3);
        drn   = (r1*rn_cached[0] + r2*rn_cached[1] + r3*rn_cached[2])/r;
        rd[0] = r1/r;
        rd[1] = r2/r;
        rd[2] = r3/r;
        
        p12 = p1*p2*det;

        for (j = 0; j < 3; ++j)
        {	for (i = 0; i < 3; ++i)
            {
                
                hei = (c3/(r*r))*(drn*(c4*delta[i][j]+3.0f*rd[i]*rd[j]) +
                    c4*(rd[j]*rn_cached[i]-rd[i]*rn_cached[j]));

                hei = hei*p12;

                helem(j, i, jg*npg + ig) = hei;
            }
        }
        __syncthreads();
    }    
	if (jg < 3 && ig < 3)
	{

		int index = 3*blockIdx.y + (3*nbe)*3*blockIdx.x + ig + (3*nbe)*jg;
		
		hest[index] = thrust::reduce(thrust::seq, &helem(jg, ig, 0), &helem(jg, ig, npg*npg));
	}
}

void cuda_ghmatece_(int* nbe,
                    int* npg,
                    int* n,
                    int* np,
                    float* c3,
                    float* c4,
                    float* fr,
                    float hestdiag[][3][3],
                    int* status
                   )
{
	dim3 threadsPerBlock(*npg,*npg);
	int shared_mem_size = 3*3*(*npg)*(*npg)*sizeof(float);
	size_t column_size = 2*(3*(*nbe))*sizeof(float);
	
	hipError_t error;

	float* device_h;

	int* device_return_status;
	int return_status;
	int width, iterations, i;

    float* hest_ = (float*) malloc(3*(*n)*3*(*nbe)*sizeof(float));
	float (*hest)[3*(*nbe)] = (float (*)[3*(*nbe)]) hest_;

	error = hipMalloc(&device_return_status, sizeof(int));
	cuda_assert(error);

	width = largest_possible_width(column_size, *n, &iterations);

	error = hipMalloc(&device_h, (3*(*nbe))*(3*(width))*sizeof(float));
	cuda_assert(error);

	error = hipMemset(device_return_status, 0, sizeof(int));
	cuda_assert(error);

	for (i = 0; i < iterations; ++i)
	{
		int starting_column = width*i;
		if (starting_column + width > *n)
			width = *n - starting_column;
		dim3 numBlocks(width, *nbe);

		error = hipMemset(device_h, 0, (3*(*nbe))*(3*(width))*sizeof(float));
		cuda_assert(error);

		hipDeviceSynchronize();
		ghmatece_kernel<<<numBlocks, threadsPerBlock, shared_mem_size>>>(
							device_cone,
							device_cx,
							device_cy,
							device_cz,
							device_cxm,
							device_cym,
							device_czm,
							device_h,
							(float (*)[3]) device_etas,
							*fr,
							device_gi,
							device_ome,
							*c3,
							*c4,
							*npg,
							*n,
							*nbe,
							starting_column,
							device_return_status
							);
		hipDeviceSynchronize();

		error = hipMemcpy(&return_status, device_return_status, sizeof(int), hipMemcpyDeviceToHost);
		cuda_assert(error);

		if (return_status != 0)
		{
			fputs("Matriz Singular\n", stderr);
		}

		error = hipMemcpy(&hest[3*starting_column], device_h, (3*(*nbe))*(3*(width))*sizeof(float), hipMemcpyDeviceToHost);
		cuda_assert(error);
	}

	error = hipFree(device_h);
	cuda_assert(error);
	*status = return_status;
	error = hipFree(device_return_status);
	cuda_assert(error);

    rigid_body_(nbe, n, hest_, hestdiag);

}
}
