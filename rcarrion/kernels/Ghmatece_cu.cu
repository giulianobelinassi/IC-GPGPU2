#include "hip/hip_runtime.h"
#include "shared.h"

#include <cstdio>
#include <cmath>
#include <hipblas.h>
#include <thrust/complex.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

extern "C"{

void rigid_body_(
            int* nbe,
            int* n,
            float hest_[],
            float hestdiag[][3][3]
        );

void rigid_body_c_(int* nbe,
		int* n,
		float hest_[],
		float hestdiag[][3][3])
{
	float (*hest)[3*(*nbe)] = (float (*)[3*(*nbe)]) hest_;
	int ma, mb, i, j, ii, jj;

	for (ma = 0; ma < *nbe; ++ma)
	{
		ii = 3*ma;
		for (mb = 0; mb < *n; ++mb)
		{
			jj = 3*mb;
			for (j = 0; j < 3; ++j)
			{
				for (i = 0; i < 3; ++i)
				{
					hestdiag[ma][j][i] -= hest[jj+j][ii+i];
				}
			}

		}
	}

}


__global__ void ghmatece_kernel(
                           int cone[],
                           float cx[],
                           float cy[],
                           float cz[],
                           float cxm[],
                           float cym[],
                           float czm[],
                           float hest[],
                           float rn[][3],
                           float fr,
                           float gi[],
                           float ome[],
                           float c3,
                           float c4,
                           int npg,
                           int n,
                           int nbe,
						   int column_pad,
                           int* ret
                           )
{

	const int ig = threadIdx.y;
	const int jg = threadIdx.x;
	const int ii = blockIdx.y;
	const int jj = blockIdx.x + column_pad;

	const int zgelem_pad = 3*3*npg*npg;
	extern __shared__ float s_[];
	
	int i, j;
	
	float p[4][2], f[4];
	float xj[3][2];
	__shared__ float co[3][4];
	__shared__ float rn_cached[4];
	float p1, p2, p12, g1, g2, sp, sm, rp, rm, det;
	float cxg, cyg, czg, cxp, cyp, czp;
	float j1, j2, j3;
	float r1, r2, r3, r, drn, rd[3];
	float hei;

//Manage the shared memory manually since apparently there is no other way
//to allocate two cubes in dynamic allocated shared memory.
//see https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
#define helem(i, j, k) s_[3*npg*npg*(i) + npg*npg*(j) + (k)]

	int iii, jjj;

	for (iii = 0; iii < 3; ++iii)
		for (jjj = 0; jjj < 3; ++jjj)
			helem(iii, jjj, npg*ig + jg) = 0;

    if (ii != jj)
    {

        if (threadIdx.x < 4 && threadIdx.y == 0)
        {
            co[0][threadIdx.x] = cx[cone[n*threadIdx.x + jj] - 1];
            co[1][threadIdx.x] = cy[cone[n*threadIdx.x + jj] - 1];
            co[2][threadIdx.x] = cz[cone[n*threadIdx.x + jj] - 1];
            rn_cached[threadIdx.x] = rn[jj][threadIdx.x];
        }
        __syncthreads();

        cxp = cxm[ii];
        cyp = cym[ii];
        czp = czm[ii];

        g2 = gi[jg];
        p2 = ome[jg];
        sp = 1 + g2;
        sm = 1 - g2;
        p[0][0] = -0.25f*sm;
        p[1][0] =  0.25f*sm;
        p[2][0] =  0.25f*sp;
        p[3][0] = -0.25f*sp;

        g1 = gi[ig];
        p1 = ome[ig];
        rp = 1 + g1;
        rm = 1 - g1;
        f[0] = 0.25f*rm*sm;
        f[1] = 0.25f*rp*sm;
        f[2] = 0.25f*rp*sp;
        f[3] = 0.25f*rm*sp;
        p[0][1] = -0.25f*rm;
        p[1][1] = -0.25f*rp;
        p[2][1] = 0.25f*rp;
        p[3][1] = 0.25f*rm;

        
       
        for (iii = 0; iii < 2; ++iii)
        {
            for (jjj = 0; jjj < 3; ++jjj)
            {
                xj[jjj][iii] = p[0][iii]*co[jjj][0] + p[1][iii]*co[jjj][1]+ p[2][iii]*co[jjj][2] + p[3][iii]*co[jjj][3];
            }
        }
        

        j1 = xj[1][0]*xj[2][1]-xj[1][1]*xj[2][0];
        j2 = xj[0][1]*xj[2][0]-xj[0][0]*xj[2][1];
        j3 = xj[0][0]*xj[1][1]-xj[0][1]*xj[1][0];

        det = sqrt(j1*j1 + j2*j2 + j3*j3);

        if (det < 1e-5)
        {
            *ret = 1;
            return;
        }


        cxg = 0;
        cyg = 0;
        czg = 0;

        for (iii = 0; iii < 4; ++iii)
        {
            cxg = cxg + co[0][iii]*f[iii];
            cyg = cyg + co[1][iii]*f[iii];
            czg = czg + co[2][iii]*f[iii];
        }

        r1    = cxg - cxp;
        r2    = cyg - cyp;
        r3    = czg - czp;

        r     = sqrt(r1*r1 + r2*r2 + r3*r3);
        drn   = (r1*rn_cached[0] + r2*rn_cached[1] + r3*rn_cached[2])/r;
        rd[0] = r1/r;
        rd[1] = r2/r;
        rd[2] = r3/r;
        
        p12 = p1*p2*det;

        for (j = 0; j < 3; ++j)
        {	for (i = 0; i < 3; ++i)
            {
                
                hei = (c3/(r*r))*(drn*(c4*delta[i][j]+3.0f*rd[i]*rd[j]) +
                    c4*(rd[j]*rn_cached[i]-rd[i]*rn_cached[j]));

                hei = hei*p12;

                helem(j, i, jg*npg + ig) = hei;
            }
        }
        __syncthreads();
    }    
	if (jg < 3 && ig < 3)
	{

		int index = 3*blockIdx.y + (3*nbe)*3*blockIdx.x + ig + (3*nbe)*jg;
		
		hest[index] = thrust::reduce(thrust::seq, &helem(jg, ig, 0), &helem(jg, ig, npg*npg));
	}
}



/*
__global__ void generate_identity(int m, float one_vec[][3])
{
	int i, j;
	const int tid = 3*(blockDim.x*blockIdx.x + threadIdx.x);
	const float const Id[3][3] = {{1., 0., 0.}, {0., 1., 0.}, {0., 0., 1.}};

	if (tid < m)
	{
		for (i = 0; i < 3; ++i)
			for (j = 0; j < 3; ++j)
				one_vec[tid + i][j] = Id[i][j];
	}
}

float* cuda_rigid_body(int nbe, int n, float device_h[])
{
	hipError_t error;
	hipblasStatus_t blaserror;
	float* device_Ids;
	float* device_hdiag;
	const int threads = 32;
	int blocks = (n+threads-1)/threads;
	dim3 threadsPerBlock(32);
	dim3 numBlocks(blocks);
	float one = 1., zero = 0.;	

	error = hipMalloc(&device_Ids, 3*3*n*sizeof(float));
	cuda_assert(error);
	error = hipMalloc(&device_hdiag, 3*3*nbe*sizeof(float));
	cuda_assert(error);

	generate_identity<<<numBlocks, threadsPerBlock>>>(n, (float (*)[3]) device_Ids);
	hipDeviceSynchronize();
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	blaserror = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nbe, 3, n, &one, device_h, nbe, device_Ids, n, 
			&zero, device_hdiag, nbe);
	cublas_assert(blaserror);

	hipFree(device_Ids);

	return device_hdiag;
}
*/

__global__ void rigid_body_kernel(int m, int n, float hest_[], float hdiag_[][3][3])
{
	int i, j, k;
	const int tid = blockDim.x*blockIdx.x + threadIdx.x;
	

#define hest(i, j) hest_[(j)*(3*m) + (i)]
#define hdiag(k, i, j) hdiag_[k][j][i]

	if (tid < m)
	{
		for (k = 0; k < n; ++k)
		{
			for (j = 0; j < 3; ++j)
			{
				int jj = 3*k + j;
				for (i = 0; i < 3; ++i)
				{
					int ii = 3*tid + i;
					hdiag(tid, i, j) -= hest(ii, jj);
				}
			}
		}
	}
#undef hest
#undef hdiag
}

float* cuda_rigid_body(int nbe, int n, float device_h[])
{
	hipError_t error;
	float* device_hdiag;
	const int threads = 128;
	int blocks = (n+threads-1)/threads;
	dim3 threadsPerBlock(threads);
	dim3 numBlocks(blocks);

	error = hipMalloc(&device_hdiag, 3*3*nbe*sizeof(float));
	cuda_assert(error);

	error = hipMemset(device_hdiag, 0, 3*3*nbe*sizeof(float));
	cuda_assert(error);

	rigid_body_kernel<<<numBlocks, threadsPerBlock>>>(nbe, n, device_h, (float (*)[3][3]) device_hdiag);

	return device_hdiag;
}


void cuda_ghmatece_(int* nbe,
                    int* npg,
                    int* n,
                    int* np,
                    float* c3,
                    float* c4,
                    float* fr,
                    float hestdiag[][3][3],
                    int* status
                   )
{
	dim3 threadsPerBlock(*npg,*npg);
	int shared_mem_size = 3*3*(*npg)*(*npg)*sizeof(float);
	size_t column_size = (3*(*nbe))*sizeof(float);
	
	hipError_t error;

	float* device_h;

	int* device_return_status;
	int return_status;
	int width, iterations, i;

    float* hest_ = (float*) malloc(3*(*n)*3*(*nbe)*sizeof(float));
	float (*hest)[3*(*nbe)] = (float (*)[3*(*nbe)]) hest_;

	error = hipMalloc(&device_return_status, sizeof(int));
	cuda_assert(error);

	width = largest_possible_width(column_size, *n, &iterations);

	error = hipMalloc(&device_h, (3*(*nbe))*(3*(width))*sizeof(float));
	cuda_assert(error);

	error = hipMemset(device_return_status, 0, sizeof(int));
	cuda_assert(error);

	for (i = 0; i < iterations; ++i)
	{
		int starting_column = width*i;
		if (starting_column + width > *n)
			width = *n - starting_column;
		dim3 numBlocks(width, *nbe);

		error = hipMemset(device_h, 0, (3*(*nbe))*(3*(width))*sizeof(float));
		cuda_assert(error);

		hipDeviceSynchronize();
		ghmatece_kernel<<<numBlocks, threadsPerBlock, shared_mem_size>>>(
							device_cone,
							device_cx,
							device_cy,
							device_cz,
							device_cxm,
							device_cym,
							device_czm,
							device_h,
							(float (*)[3]) device_etas,
							*fr,
							device_gi,
							device_ome,
							*c3,
							*c4,
							*npg,
							*n,
							*nbe,
							starting_column,
							device_return_status
							);
		hipDeviceSynchronize();

		error = hipMemcpy(&return_status, device_return_status, sizeof(int), hipMemcpyDeviceToHost);
		cuda_assert(error);

		if (return_status != 0)
		{
			fputs("Matriz Singular\n", stderr);
		}

//		error = hipMemcpy(&hest[3*starting_column], device_h, (3*(*nbe))*(3*(width))*sizeof(float), hipMemcpyDeviceToHost);
//		cuda_assert(error);
	}

	float* device_hdiag = cuda_rigid_body(*nbe, *n, device_h);

	error = hipFree(device_h);
	cuda_assert(error);
	*status = return_status;
	error = hipFree(device_return_status);
	cuda_assert(error);

	/*Guarda em shared.cu*/
	device_hestdiag = device_hdiag;

#ifdef TEST_CUDA
	error = hipMemcpy(hestdiag, device_hdiag, 3*3*(*nbe)*sizeof(float), hipMemcpyDeviceToHost);
	cuda_assert(error);
#endif
//    rigid_body_(nbe, n, hest_, hestdiag);

}

void cuda_send_gest_data_(int* nbe, float* gestdiag)
{
	hipError_t error;
	error = hipMalloc(&device_gestdiag, (*nbe)*3*3*sizeof(float));
	cuda_assert(error);
	error = hipMemcpy(device_gestdiag, gestdiag, (*nbe)*3*3*sizeof(float), hipMemcpyHostToDevice);
	cuda_assert(error);
}

}
