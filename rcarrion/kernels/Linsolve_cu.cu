#include "shared.h"
#include "../include/magma.h"

#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C"{

void lu_assert(int status)
{
    if (status != 0)
    {
        fputs("ERRO: Matriz ZH singular\n", stderr);
        exit(1);
    }
}

void cuda_linsolve_(
			int* nn,
			int* n,
			thrust::complex<float> zh[],
			thrust::complex<float> zfi[]
		)
{
//	hipblasHandle_t handle;
//	hipblasStatus_t stats;
    hipError_t error;

    thrust::complex<float>* device_zh;
    thrust::complex<float>* device_zfi;
//	hipComplex** device_matrix_pointer;

//    int* device_piv;
//    int* device_status;
	int status;
	int* piv = (int*) malloc((*nn)*sizeof(int));


    thrust::complex<float> one(1.0f, 0.0f);

	error = hipMalloc(&device_zh, (*nn)*(3*(*n))*sizeof(thrust::complex<float>));
	cuda_assert(error);

    error = hipMalloc(&device_zfi, (*nn)*sizeof(thrust::complex<float>));
    cuda_assert(error);

//    error = hipMalloc(&device_piv, (*nn)*sizeof(int));
//   cuda_assert(error);
	
//	error = hipMalloc(&device_matrix_pointer, sizeof(hipComplex*));
//	cuda_assert(error);

	error = hipMemcpy(device_zh, zh, (*nn)*(3*(*n))*sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_zfi, zfi, (*nn)*sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
	cuda_assert(error);

	magma_init();
	magma_cgesv_gpu(*nn, 1, (magmaFloatComplex_ptr) device_zh, *nn, piv, (magmaFloatComplex_ptr) device_zfi, *nn, &status);
	magma_finalize();

//	error = hipMemcpy(device_matrix_pointer, (hipComplex*) &device_zh, sizeof(thrust::complex<float>*), hipMemcpyHostToDevice);
//	cuda_assert(error);

//    error = hipMalloc(&device_status, sizeof(int));
//    cuda_assert(error);

//    stats = hipblasCreate(&handle);
//	cublas_assert(stats);

//	stats = hipblasCgetrfBatched(handle, *nn, (hipComplex**) device_matrix_pointer, *nn, device_piv, device_status, 1);
//    hipDeviceSynchronize();
//    cublas_assert(stats);

//    error = hipMemcpy(&status, device_status, sizeof(int), hipMemcpyDeviceToHost);
//    cuda_assert(error);
//    lu_assert(status);
    
//    stats = hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, *nn, 1, (hipComplex*) &one, (hipComplex*) device_zh, *nn, (hipComplex*) device_zfi, *nn);
//   hipDeviceSynchronize();
//    cublas_assert(stats);

//    stats = hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, *nn, 1, (hipComplex*) &one, (hipComplex*) device_zh, *nn, (hipComplex*) device_zfi, *nn);
//    hipDeviceSynchronize();
 //   cublas_assert(stats);

	error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
    cuda_assert(error);

    error = hipFree(device_zh);
    cuda_assert(error);

//    error = hipFree(device_status);
//    cuda_assert(error);

    error = hipFree(device_zfi);
    cuda_assert(error);

//    error = hipFree(device_piv);
//    cuda_assert(error);

//	error = hipFree(device_matrix_pointer);
//	cuda_assert(error);

	free(piv);
//	hipblasDestroy(handle);
//
}
}
