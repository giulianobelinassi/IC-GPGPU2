#include "shared.h"
#include "../include/magma.h"

#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C"{

void lu_assert(int status)
{
    if (status != 0)
    {
        fputs("ERRO: Matriz ZH singular\n", stderr);
        exit(1);
    }
}

void cuda_linsolve_(
			int* nn,
			int* n,
			thrust::complex<FREAL> zfi[],
            thrust::complex<FREAL> zdfi[]
		)
{
    hipError_t error;

	thrust::complex<FREAL>* device_zfi;
    thrust::complex<FREAL>* device_zdfi;

	int status;
	int* piv = (int*) malloc((*nn)*sizeof(int));

    thrust::complex<FREAL> one(1.0f, 0.0f);
    thrust::complex<FREAL> zero(0., 0.);
    
    hipblasHandle_t handle;
	hipblasStatus_t stats;


	if (!piv)
	{
		fputs("Erro: Matriz ZH singular", stderr);
		exit(1);
	}

	magma_init();

    error = hipMalloc(&device_zfi, (*nn)*sizeof(thrust::complex<FREAL>));
    cuda_assert(error);

    error = hipMalloc(&device_zdfi, (*nn)*sizeof(thrust::complex<FREAL>));
    cuda_assert(error);

	error = hipMemcpy(device_zdfi, zdfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyHostToDevice);
	cuda_assert(error);


    stats = hipblasCreate(&handle);
	cublas_assert(stats);

    if (sizeof(FREAL) == 8)
        stats = hipblasZgemv(handle, HIPBLAS_OP_N, (*nn), (*nn), (hipDoubleComplex*) &one, (hipDoubleComplex*) device_zg, (*nn), (hipDoubleComplex*) device_zdfi, 1, (hipDoubleComplex*) &zero, (hipDoubleComplex*) device_zfi, 1); 
    else
        stats = hipblasCgemv(handle, HIPBLAS_OP_N, (*nn), (*nn), (hipComplex*) &one, (hipComplex*) device_zg, (*nn), (hipComplex*) device_zdfi, 1, (hipComplex*) &zero, (hipComplex*) device_zfi, 1); 
    cublas_assert(stats);
	hipblasDestroy(handle);

    error = hipFree(device_zg);
    cuda_assert(error);
    error = hipFree(device_zdfi);
    cuda_assert(error);
    
    if (sizeof(FREAL) == 8)
		magma_zgesv_gpu(*nn, 1, (magmaDoubleComplex_ptr) device_zh, *nn, piv, (magmaDoubleComplex_ptr) device_zfi, *nn, &status);
	else
		magma_cgesv_gpu(*nn, 1, (magmaFloatComplex_ptr) device_zh, *nn, piv, (magmaFloatComplex_ptr) device_zfi, *nn, &status);
	
	magma_finalize();
	lu_assert(status);

	error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
    cuda_assert(error);

    error = hipFree(device_zh);
    cuda_assert(error);

    error = hipFree(device_zfi);
    cuda_assert(error);

	free(piv);
}
}
