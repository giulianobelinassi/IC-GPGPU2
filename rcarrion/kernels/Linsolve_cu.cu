#include "shared.h"
#include "../include/magma.h"

#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C"{

void lu_assert(int status)
{
    if (status != 0)
    {
        fputs("ERRO: Matriz ZH singular\n", stderr);
        exit(1);
    }
}

void cuda_linsolve_(
			int* nn,
			int* n,
			thrust::complex<FREAL> zfi[]
		)
{
    hipError_t error;

	int status;
	int* piv = (int*) malloc((*nn)*sizeof(int));

    thrust::complex<FREAL> one(1.0f, 0.0f);
    thrust::complex<FREAL> zero(0., 0.);
    
	if (!piv)
	{
		fputs("Erro: Matriz ZH singular", stderr);
		exit(1);
	}

	magma_init();

    if (sizeof(FREAL) == 8)
		magma_zgesv_gpu(*nn, 1, (magmaDoubleComplex_ptr) device_zh, *nn, piv, (magmaDoubleComplex_ptr) device_zfi, *nn, &status);
	else
		magma_cgesv_gpu(*nn, 1, (magmaFloatComplex_ptr) device_zh, *nn, piv, (magmaFloatComplex_ptr) device_zfi, *nn, &status);
	
	magma_finalize();
	lu_assert(status);

	error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
    cuda_assert(error);

    error = hipFree(device_zh);
    cuda_assert(error);

    error = hipFree(device_zfi);
    cuda_assert(error);

	free(piv);
}
}
