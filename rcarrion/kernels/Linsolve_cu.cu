#include "shared.h"
#include "../include/magma.h"

#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C"{

void lu_assert(int status)
{
    if (status != 0)
    {
        fputs("ERRO: Matriz ZH singular\n", stderr);
        exit(1);
    }
}

static void reorganize_zh(int nn, thrust::complex<FREAL> zh[])
{
    hipError_t error;
    size_t zh_size = nn*nn*sizeof(thrust::complex<FREAL>);

    error = hipFree(device_zh);
    cuda_assert(error);

    error = hipMalloc(&device_zh, zh_size);
    cuda_assert(error);

    error = hipMemcpy(device_zh, zh, zh_size, hipMemcpyHostToDevice);
    cuda_assert(error);
}

static int zh_fits_in_memory(int nn, thrust::complex<FREAL> zh[])
{
    hipError_t error;
    size_t zh_size = nn*nn*sizeof(thrust::complex<FREAL>);
    size_t available_mem;
    size_t total_mem;

    error = hipMemGetInfo(&available_mem, &total_mem);
    cuda_assert(error);

    if (available_mem < zh_size)
        return 0;
    return 1;

}

/*Fortran routine coded in Linsolve.for*/
void linsolve_cpu_(int* nn, thrust::complex<FREAL> zh[], thrust::complex<FREAL> zfi[]);

void cuda_linsolve_(
			int* nn,
			int* n,
            thrust::complex<FREAL> zh[],
			thrust::complex<FREAL> zfi[]
		)
{
    hipError_t error;

	int status;

    thrust::complex<FREAL> one(1.0f, 0.0f);
    thrust::complex<FREAL> zero(0., 0.);
    

    if (zh_fits_in_memory(*nn, zh))
    {
	    int* piv = (int*) malloc((*nn)*sizeof(int));
        if (swapped)
            reorganize_zh(*nn, zh);

		if (!piv)
		{
			fputs("Erro: Memória Insuficiente", stderr);
			exit(1);
		}

        magma_init();

        if (sizeof(FREAL) == 8)
            magma_zgesv_gpu(*nn, 1, (magmaDoubleComplex_ptr) device_zh, *nn, piv, (magmaDoubleComplex_ptr) device_zfi, *nn, &status);
        else
            magma_cgesv_gpu(*nn, 1, (magmaFloatComplex_ptr) device_zh, *nn, piv, (magmaFloatComplex_ptr) device_zfi, *nn, &status);
        
        magma_finalize();
        lu_assert(status);

        error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
        cuda_assert(error);

	    free(piv);
    }
    else
    {
        error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
        cuda_assert(error);
        
        linsolve_cpu_(nn, zh, zfi);
    }

    error = hipFree(device_zh);
    cuda_assert(error);

    error = hipFree(device_zfi);
    cuda_assert(error);
}
}
