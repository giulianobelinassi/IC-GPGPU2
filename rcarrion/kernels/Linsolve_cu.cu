#include "shared.h"

#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C"{

void lu_assert(int status)
{
    if (status != 0)
    {
        fputs("ERRO: Matriz ZH singular\n", stderr);
        exit(1);
    }
}

void cgetrf_(int* m, int* n, thrust::complex<float> A[], int* lda, int piv[], int* info);

void cuda_linsolve_(
			int* nn,
			int* n,
			thrust::complex<float> zh[],
			thrust::complex<float> zfi[]
		)
{
	hipblasHandle_t handle;
	hipblasStatus_t stats;
    hipError_t error;

    thrust::complex<float>* device_zh;
    thrust::complex<float>* device_zfi;
	hipComplex** device_matrix_pointer;

    int* device_piv;
    int* device_status;
	int status;

    thrust::complex<float> one(1.0f, 0.0f);

	thrust::complex<float> zhp[(*nn)*(*n)*3];
	int piv[(*nn)*(*n)*3];

	error = hipMalloc(&device_zh, (*nn)*(3*(*n))*sizeof(thrust::complex<float>));
	cuda_assert(error);

    error = hipMalloc(&device_zfi, (*nn)*sizeof(thrust::complex<float>));
    cuda_assert(error);

    error = hipMalloc(&device_piv, (*nn)*sizeof(int));
    cuda_assert(error);

//    error = hipMemset(device_piv, 0, (*nn)*sizeof(int));
//    cuda_assert(error);

	error = hipMalloc(&device_matrix_pointer, sizeof(hipComplex*));
	cuda_assert(error);

	error = hipMemcpy(device_zh, zh, (*nn)*(3*(*n))*sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_zfi, zfi, (*nn)*sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_matrix_pointer, (hipComplex*) &device_zh, sizeof(thrust::complex<float>*), hipMemcpyHostToDevice);
	cuda_assert(error);

    error = hipMalloc(&device_status, sizeof(int));
    cuda_assert(error);

    stats = hipblasCreate(&handle);
	cublas_assert(stats);

	stats = hipblasCgetrfBatched(handle, *nn, (hipComplex**) device_matrix_pointer, *nn, device_piv, device_status, 1);
    hipDeviceSynchronize();
    cublas_assert(stats);

	error = hipMemcpy(zhp, device_zh, (*nn)*(3*(*n))*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
	cuda_assert(error);

//	cgetrf_(nn, nn, zh, nn, piv, &status);

    error = hipMemcpy(&status, device_status, sizeof(int), hipMemcpyDeviceToHost);
    cuda_assert(error);
    lu_assert(status);
    
    stats = hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, *nn, *nn, (hipComplex*) &one, (hipComplex*) device_zh, *nn, (hipComplex*) device_zfi, *nn);
    hipDeviceSynchronize();
    cublas_assert(stats);

    stats = hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, *nn, *nn, (hipComplex*) &one, (hipComplex*) device_zh, *nn, (hipComplex*) device_zfi, *nn);
    hipDeviceSynchronize();
    cublas_assert(stats);

    error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
    cuda_assert(error);

    error = hipFree(device_zh);
    cuda_assert(error);

    error = hipFree(device_status);
    cuda_assert(error);

    error = hipFree(device_zfi);
    cuda_assert(error);

    error = hipFree(device_piv);
    cuda_assert(error);

	error = hipFree(device_matrix_pointer);
	cuda_assert(error);

	hipblasDestroy(handle);

}
}
