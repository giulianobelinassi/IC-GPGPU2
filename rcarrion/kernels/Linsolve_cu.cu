#include "shared.h"
#include "../include/magma.h"

#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C"{

void lu_assert(int status)
{
    if (status != 0)
    {
        fputs("ERRO: Matriz ZH singular\n", stderr);
        exit(1);
    }
}

void cuda_linsolve_(
			int* nn,
			int* n,
			thrust::complex<FREAL> zh[],
			thrust::complex<FREAL> zfi[]
		)
{
    hipError_t error;

	thrust::complex<FREAL>* device_zh;
	thrust::complex<FREAL>* device_zfi;

	int status;
	int* piv = (int*) malloc((*nn)*sizeof(int));

    thrust::complex<FREAL> one(1.0f, 0.0f);

	if (!piv)
	{
		fputs("Erro: Matriz ZH singular", stderr);
		exit(1);
	}

	magma_init();

	error = hipMalloc(&device_zh, (*nn)*(*nn)*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

    error = hipMalloc(&device_zfi, (*nn)*sizeof(thrust::complex<FREAL>));
    cuda_assert(error);

	error = hipMemcpy(device_zh, zh, (*nn)*(*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_zfi, zfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyHostToDevice);
	cuda_assert(error);

	if (sizeof(FREAL) == 8)
		magma_zgesv_gpu(*nn, 1, (magmaDoubleComplex_ptr) device_zh, *nn, piv, (magmaDoubleComplex_ptr) device_zfi, *nn, &status);
	else
		magma_cgesv_gpu(*nn, 1, (magmaFloatComplex_ptr) device_zh, *nn, piv, (magmaFloatComplex_ptr) device_zfi, *nn, &status);
	
	magma_finalize();
	lu_assert(status);

	error = hipMemcpy(zfi, device_zfi, (*nn)*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
    cuda_assert(error);

    error = hipFree(device_zh);
    cuda_assert(error);

    error = hipFree(device_zfi);
    cuda_assert(error);

	free(piv);
}
}
