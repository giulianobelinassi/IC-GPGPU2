#include "hip/hip_runtime.h"
#include "shared.h"

#include <cstdio>
#include <cmath>
#include <thrust/complex.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

extern "C"{

__global__ void ghmatecd_kernel(
                           int cone[],
                           FREAL cx[],
                           FREAL cy[],
                           FREAL cz[],
                           FREAL cxm[],
                           FREAL cym[],
                           FREAL czm[],
                           thrust::complex<FREAL> zh[],
                           thrust::complex<FREAL> zg[],
                           FREAL rn[][3],
                           thrust::complex<FREAL> zge,
                           thrust::complex<FREAL> zcs,
                           thrust::complex<FREAL> zcp,
                           FREAL fr,
                           FREAL gi[],
                           FREAL ome[],
                           FREAL c1,
                           FREAL c2,
                           FREAL c3,
                           FREAL c4,
//                           int npg,
                           int n,
                           int nbe,
                           int dim_cone,
						   int column_pad,
                           int fast_singular,
                           int* ret
                           )
{

	const int ig = threadIdx.y;
	const int jg = threadIdx.x;
	const int ii = blockIdx.y;
	const int jj = blockIdx.x + column_pad;

	const int npg = blockDim.y;

	const int tid = npg*threadIdx.y + threadIdx.x;
	const int lane = tid % 32;
	const int warp = tid / 32;
	const int num_warps = (npg*npg + 31)/32;

	const int lane_x = lane % 3;
	const int lane_y = lane / 3;

	const int gelem_pad = 3*3*num_warps;
	extern __shared__ thrust::complex<FREAL> s_[];
	
	int i, j;
	
	const FREAL pi  = 3.141592654;
	FREAL p[4][2], f[4];
	FREAL xj[3][2];
	__shared__ FREAL co[3][4];
	__shared__ FREAL rn_cached[4];
	FREAL g1, g2, p1, p2, p12, sp, sm, rp, rm, det;
	FREAL cxg, cyg, czg, cxp, cyp, czp;
	FREAL j1, j2, j3;
	FREAL r1, r2, r3, r, drn, rd[3];
	thrust::complex<FREAL> zwi, zc0, zc1, zc2, zkp, zks, zzp, zzs, zezp, zezs, 
                    zp2, zs2, zfhi, zcappa, zfhidr, zcappadr, zaa, zbb, zcc;

	thrust::complex<FREAL> zhi, zgi;

	float zhi_real, zhi_imag, zgi_real, zgi_imag;

	int iii, jjj;


#define zhelem(i, j, k) s_[3*num_warps*(i) + num_warps*(j) + (k)]
#define zgelem(i, j, k) (s_ + gelem_pad)[3*num_warps*(i) + num_warps*(j) + (k)]
	
	
	if (threadIdx.x < 4 && threadIdx.y == 0)
	{
		co[0][threadIdx.x] = cx[cone[dim_cone*threadIdx.x + jj] - 1];
		co[1][threadIdx.x] = cy[cone[dim_cone*threadIdx.x + jj] - 1];
		co[2][threadIdx.x] = cz[cone[dim_cone*threadIdx.x + jj] - 1];

		//Note que a dimensão coluna de rn é 3, mas estamos acessando o elemento
		//na posição 4. Isto pode levar a um segfault, entretanto consegue-se
		//uma melhora de ~100ms no kernel se fizermos esta alteração.
		rn_cached[threadIdx.x] = rn[jj][threadIdx.x];
	}
	__syncthreads();

	cxp = cxm[ii];
	cyp = cym[ii];
	czp = czm[ii];

	g2 = gi[jg];
	p2 = ome[jg];
	sp = 1 + g2;
	sm = 1 - g2;
	p[0][0] = -0.25*sm;
	p[1][0] =  0.25*sm;
	p[2][0] =  0.25*sp;
	p[3][0] = -0.25*sp;

	g1 = gi[ig];
	p1 = ome[ig];
	rp = 1 + g1;
	rm = 1 - g1;
	f[0] = 0.25*rm*sm;
	f[1] = 0.25*rp*sm;
	f[2] = 0.25*rp*sp;
	f[3] = 0.25*rm*sp;
	p[0][1] = -0.25*rm;
	p[1][1] = -0.25*rp;
	p[2][1] = 0.25*rp;
	p[3][1] = 0.25*rm;

	
   
	for (iii = 0; iii < 2; ++iii)
	{
		for (jjj = 0; jjj < 3; ++jjj)
		{
			xj[jjj][iii] = p[0][iii]*co[jjj][0] + p[1][iii]*co[jjj][1]+ p[2][iii]*co[jjj][2] + p[3][iii]*co[jjj][3];
		}
	}
    

    j1 = xj[1][0]*xj[2][1]-xj[1][1]*xj[2][0];
	j2 = xj[0][1]*xj[2][0]-xj[0][0]*xj[2][1];
	j3 = xj[0][0]*xj[1][1]-xj[0][1]*xj[1][0];

	det = sqrt(j1*j1 + j2*j2 + j3*j3);

	if (det < 1e-5)
	{
		*ret = 1;
		return;
	}


	cxg = 0;
	cyg = 0;
	czg = 0;

	for (iii = 0; iii < 4; ++iii)
	{
		cxg = cxg + co[0][iii]*f[iii];
		cyg = cyg + co[1][iii]*f[iii];
		czg = czg + co[2][iii]*f[iii];
	}

	r1    = cxg - cxp;
	r2    = cyg - cyp;
	r3    = czg - czp;

	r     = sqrt(r1*r1 + r2*r2 + r3*r3);
	drn   = (r1*rn_cached[0] + r2*rn_cached[1] + r3*rn_cached[2])/r;
	rd[0] = r1/r;
	rd[1] = r2/r;
	rd[2] = r3/r;
	

	zwi = thrust::complex<FREAL>(0, fr);
	
	zc0 = ((FREAL) 1.0)/(((FREAL) 4.)*(pi)*(zge));
	zc1 = ((zcp)/(zcs))*((zcp)/(zcs));
	zc2 = ((zcs)/(zcp))*((zcs)/(zcp));
	zkp = -zwi/(zcp);
	zks = -zwi/(zcs);
	zzp = zkp*r;
	zzs = zks*r;
	zezp= exp(zzp);
	zezs= exp(zzs);
	zp2 = zzp*zzp;
	zs2 = zzs*zzs;

	zfhi    = (((FREAL) 1.) + ((FREAL) 1.)/zs2 - ((FREAL) 1.)/zzs)*zezs/r - zc2*(((FREAL) 1.)/zp2 - ((FREAL) 1.)/zzp)*zezp/r;
	zcappa  = (((FREAL) 1.) + ((FREAL) 3.)/zs2 - ((FREAL) 3.f)/zzs)*zezs/r - zc2*(((FREAL) 1.) + ((FREAL) 3.)/zp2 - ((FREAL) 3.)/zzp)*zezp/r;
	zfhidr  = (((FREAL) -2.)+ zzs + ((FREAL) 3.)/zzs - ((FREAL) 3.)/zs2)*zezs/(r*r) - zc2*(((FREAL) -1.) + ((FREAL) 3.)/zzp - ((FREAL) 3.)/zp2)*zezp/(r*r);
	zcappadr= (zzs - ((FREAL) 4.) + ((FREAL) 9.f)/zzs - ((FREAL) 9.)/zs2)*zezs/(r*r) - zc2*(zzp - ((FREAL) 4.) + ((FREAL) 9.)/zzp - ((FREAL) 9.f)/zp2)*zezp/(r*r);

	zaa = zfhidr-zcappa/r;
	zbb = ((FREAL) 4.)*zcappa/r -((FREAL) 2.)*zcappadr;
	zcc = (zc1-((FREAL) 2.))*(zaa + ((FREAL) 0.5)*zbb-((FREAL) 3.0)*zcappa/r)-((FREAL) 2.0)*zcappa/r;

	p12 = p1*p2*det;
	
    for (j = 0; j < 3; ++j)
    {	for (i = 0; i < 3; ++i)
        {
            zgi = (zc0*(zfhi*delta[j][i] - zcappa*rd[j]*rd[i]));
            

            zhi = (((FREAL) 1.0)/(((FREAL) 4.0)*pi))*((zaa*(drn*delta[j][i] + 
                                rd[j]*rn_cached[i])) + rd[i]*rd[j]*drn*zbb + 
                        rd[i]*rn_cached[j]*zcc);
        
            if (ii == jj && fast_singular)
            {
                zgi = zgi - (c1/r)*(c2*delta[j][i] + rd[i]*rd[j]);
                zhi = zhi - (c3/(r*r))*(drn*(c4*delta[j][i] + ((FREAL) 3.0)*rd[i]*rd[j]) + c4*(rd[j]*rn_cached[i] - rd[i]*rn_cached[j]));
            }
          
            zgi = zgi*p12;
			zgi_real = zgi.real();
			zgi_imag = zgi.imag();

			for (int offset = 16; offset > 0; offset = offset/2)
				zgi_real += __shfl_down(zgi_real, offset);
			for (int offset = 16; offset > 0; offset = offset/2)
				zgi_imag += __shfl_down(zgi_imag, offset);
			
            zhi = zhi*p12;
			zhi_real = zhi.real();
			zhi_imag = zhi.imag();
			for (int offset = 16; offset > 0; offset = offset/2)
				zhi_real += __shfl_down(zhi_real, offset);
			for (int offset = 16; offset > 0; offset = offset/2)
				zhi_imag += __shfl_down(zhi_imag, offset);

			if (lane == 0)
			{
				zhelem(j, i, warp) = thrust::complex<float>(zhi_real, zhi_imag);
				zgelem(j, i, warp) = thrust::complex<float>(zgi_real, zgi_imag);
			}
        }
    }
	__syncthreads();
	
	if (jg < 3 && ig < 3)
	{
		int index = 3*blockIdx.y + (3*nbe)*3*blockIdx.x + ig + (3*nbe)*jg;
		zg[index] = thrust::reduce(thrust::seq, &zgelem(jg, ig, 0), &zgelem(jg, ig, num_warps));
	} else if ((npg-3) <= jg && jg < npg && (npg-3) <= ig && ig < npg) //Split the warps
	{
		int index = 3*blockIdx.y + (3*nbe)*3*blockIdx.x + (ig-(npg-3)) + (3*nbe)*(jg-(npg-3));
		zh[index] = thrust::reduce(thrust::seq, &zhelem((jg-(npg-3)), (ig-(npg-3)), 0), &zhelem((jg-(npg-3)), (ig-(npg-3)), num_warps));
	}

//	} else if (3 <= jg && jg  < 6 && 3 <= ig && ig < 6)
//	{
//		int index = 3*blockIdx.y + (3*nbe)*3*blockIdx.x + (ig-3) + (3*nbe)*(jg-3);
//		zh[index] = thrust::reduce(thrust::seq, &zhelem((jg-3), (ig-3), 0), &zhelem((jg-3), (ig-3), npg*npg));
//	}
}


void cuda_ghmatecd_(int* nbe,
                    int* npg,
                    int* n,
                    int* np,
                    thrust::complex<FREAL>* zge,
                    thrust::complex<FREAL>* zcs,
                    thrust::complex<FREAL>* zcp,
                    FREAL* c1,
                    FREAL* c2,
                    FREAL* c3,
                    FREAL* c4,
                    FREAL* fr,
                    FREAL* zhest_,
                    FREAL* zgest_,
                    thrust::complex<FREAL>* zgp_,
                    thrust::complex<FREAL>* zhp_,
                    int* fast_singular,
                    int* status
                   )
{
	
	size_t column_size = 2*(3*(*nbe))*sizeof(thrust::complex<FREAL>);
	
	int shared_mem_size = 2*3*3*(*npg)*(*npg)*sizeof(thrust::complex<FREAL>);
	hipError_t error;
	
	thrust::complex<FREAL>* device_zh;
	thrust::complex<FREAL>* device_zg;

	int* device_return_status;
	int return_status;

	/*Cast os parâmetros de volta para o tipo original*/
//	FREAL (*zhest)[3*(*nbe)] = (FREAL (*)[3*(*nbe)]) zhest_;
//	FREAL (*zgest)[3*(*nbe)] = (FREAL (*)[3*(*nbe)]) zgest_;
	thrust::complex<FREAL> (*zgp)[3*(*nbe)] = (thrust::complex<FREAL> (*)[3*(*nbe)]) zgp_;
	thrust::complex<FREAL> (*zhp)[3*(*nbe)] = (thrust::complex<FREAL> (*)[3*(*nbe)]) zhp_;

	int i, iterations, width;
	dim3 threadsPerBlock(*npg,*npg);

	error = hipMalloc(&device_return_status, sizeof(int));
	cuda_assert(error);

	width = largest_possible_width(column_size, *nbe, &iterations);

	error = hipMalloc(&device_zh, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	error = hipMalloc(&device_zg, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<FREAL>));
	cuda_assert(error);

	for (i = 0; i < iterations; ++i)
	{
		int starting_column = width*i;
//		if (starting_column + width > *n)
//			width = *n - starting_column;
		if (starting_column + width > *nbe)
			width = *nbe - starting_column;
		dim3 numBlocks(width, *nbe);


		error = hipMemset(device_return_status, 0, sizeof(int));
		cuda_assert(error);

		error = hipMemset(device_zh, 0, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<FREAL>));
		cuda_assert(error);

		error = hipMemset(device_zg, 0, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<FREAL>));
		cuda_assert(error);

		hipDeviceSynchronize();
		ghmatecd_kernel<<<numBlocks, threadsPerBlock, shared_mem_size>>>(
							device_cone,
							device_cx,
							device_cy,
							device_cz,
							device_cxm,
							device_cym,
							device_czm,
							device_zh,
							device_zg,
							(FREAL (*)[3]) device_etas,
							*zge,
							*zcs,
							*zcp,
							*fr,
							device_gi,
							device_ome,
							*c1,
							*c2,
							*c3,
							*c4,
//							*npg,
							*n,
							*nbe,
							*n,
							starting_column,
                            *fast_singular,
							device_return_status
							);
		hipDeviceSynchronize();

		error = hipMemcpy(&return_status, device_return_status, sizeof(int), hipMemcpyDeviceToHost);
		cuda_assert(error);

		if (return_status != 0)
		{
			fputs("Matriz Singular\n", stderr);
		}

		error = hipMemcpy(&zhp[3*starting_column], device_zh, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
		cuda_assert(error);
		error = hipMemcpy(&zgp[3*starting_column], device_zg, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<FREAL>), hipMemcpyDeviceToHost);
		cuda_assert(error);

	}

	error = hipFree(device_zh);
	cuda_assert(error);
	error = hipFree(device_zg);
	cuda_assert(error);
	*status = return_status;
    error = hipFree(device_return_status);
    cuda_assert(error);
}
}
