#include "hip/hip_runtime.h"
#include "shared.h"

#include <cstdio>
#include <cmath>
#include <thrust/complex.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

extern "C"{

__global__ void ghmatecd_kernel(
                           int cone[],
                           float cx[],
                           float cy[],
                           float cz[],
                           float cxm[],
                           float cym[],
                           float czm[],
                           thrust::complex<float> zh[],
                           thrust::complex<float> zg[],
                           float rn[][3],
                           thrust::complex<float> zge,
                           thrust::complex<float> zcs,
                           thrust::complex<float> zcp,
                           float fr,
                           float gi[],
                           float ome[],
                           float c1,
                           float c2,
                           float c3,
                           float c4,
                           int npg,
                           int n,
                           int nbe,
                           int interec,
                           int dim_cone,
						   int column_pad,
                           int* ret
                           )
{

	const int ig = threadIdx.y;
	const int jg = threadIdx.x;
	const int ii = blockIdx.y;
	const int jj = blockIdx.x + column_pad;

	const int gelem_pad = 3*3*npg*npg;
	extern __shared__ thrust::complex<float> s_[];
	
	int i, j;
	
	const float pi  = 3.141592654;
	float p[4][2], f[4];
	float xj[3][2];
	__shared__ float co[3][4];
	__shared__ float rn_cached[4];
	float g1, g2, p1, p2, p12, sp, sm, rp, rm, det;
	float cxg, cyg, czg, cxp, cyp, czp;
	float j1, j2, j3;
	float r1, r2, r3, r, drn, rd[3];
	thrust::complex<float> zwi, zc0, zc1, zc2, zkp, zks, zzp, zzs, zezp, zezs, 
                    zp2, zs2, zfhi, zcappa, zfhidr, zcappadr, zaa, zbb, zcc;

	thrust::complex<float> zhi, zgi;

	int iii, jjj;

#define zhelem(i, j, k) s_[3*npg*npg*(i) + npg*npg*(j) + (k)]
#define zgelem(i, j, k) (s_ + gelem_pad)[3*npg*npg*(i) + npg*npg*(j) + (k)]

	zgelem(0, 0, npg*ig + jg) = 0;
	zgelem(0, 1, npg*ig + jg) = 0;
	zgelem(0, 2, npg*ig + jg) = 0;
	zgelem(1, 0, npg*ig + jg) = 0;
	zgelem(1, 1, npg*ig + jg) = 0;
	zgelem(1, 2, npg*ig + jg) = 0;
	zgelem(2, 0, npg*ig + jg) = 0;
	zgelem(2, 1, npg*ig + jg) = 0;
	zgelem(2, 2, npg*ig + jg) = 0;

	zhelem(0, 0, npg*ig + jg) = 0;
	zhelem(0, 1, npg*ig + jg) = 0;
	zhelem(0, 2, npg*ig + jg) = 0;
	zhelem(1, 0, npg*ig + jg) = 0;
	zhelem(1, 1, npg*ig + jg) = 0;
	zhelem(1, 2, npg*ig + jg) = 0;
	zhelem(2, 0, npg*ig + jg) = 0;
	zhelem(2, 1, npg*ig + jg) = 0;
	zhelem(2, 2, npg*ig + jg) = 0;
	
	if (threadIdx.x < 4 && threadIdx.y == 0)
	{
		co[0][threadIdx.x] = cx[cone[dim_cone*threadIdx.x + jj] - 1];
		co[1][threadIdx.x] = cy[cone[dim_cone*threadIdx.x + jj] - 1];
		co[2][threadIdx.x] = cz[cone[dim_cone*threadIdx.x + jj] - 1];
		//Note que a dimensão coluna de rn é 3, mas estamos acessando o elemento
		//na posição 4. Isto pode levar a um segfault, entretanto consegue-se
		//uma melhora de ~100ms no kernel se fizermos esta alteração.
		rn_cached[threadIdx.x] = rn[jj][threadIdx.x];
	}
	__syncthreads();

	cxp = cxm[ii];
	cyp = cym[ii];
	czp = czm[ii];

	g2 = gi[jg];
	p2 = ome[jg];
	sp = 1 + g2;
	sm = 1 - g2;
	p[0][0] = -0.25f*sm;
	p[1][0] =  0.25f*sm;
	p[2][0] =  0.25f*sp;
	p[3][0] = -0.25f*sp;

	g1 = gi[ig];
	p1 = ome[ig];
	rp = 1 + g1;
	rm = 1 - g1;
	f[0] = 0.25f*rm*sm;
	f[1] = 0.25f*rp*sm;
	f[2] = 0.25f*rp*sp;
	f[3] = 0.25f*rm*sp;
	p[0][1] = -0.25f*rm;
	p[1][1] = -0.25f*rp;
	p[2][1] = 0.25f*rp;
	p[3][1] = 0.25f*rm;

	
   
	for (iii = 0; iii < 2; ++iii)
	{
		for (jjj = 0; jjj < 3; ++jjj)
		{
			xj[jjj][iii] = p[0][iii]*co[jjj][0] + p[1][iii]*co[jjj][1]+ p[2][iii]*co[jjj][2] + p[3][iii]*co[jjj][3];
		}
	}
    

    j1 = xj[1][0]*xj[2][1]-xj[1][1]*xj[2][0];
	j2 = xj[0][1]*xj[2][0]-xj[0][0]*xj[2][1];
	j3 = xj[0][0]*xj[1][1]-xj[0][1]*xj[1][0];

	det = sqrt(j1*j1 + j2*j2 + j3*j3);

	if (det < 1e-5)
	{
		*ret = 1;
		return;
	}


	cxg = 0;
	cyg = 0;
	czg = 0;

	for (iii = 0; iii < 4; ++iii)
	{
		cxg = cxg + co[0][iii]*f[iii];
		cyg = cyg + co[1][iii]*f[iii];
		czg = czg + co[2][iii]*f[iii];
	}

	r1    = cxg - cxp;
	r2    = cyg - cyp;
	r3    = czg - czp;

	r     = sqrt(r1*r1 + r2*r2 + r3*r3);
	drn   = (r1*rn_cached[0] + r2*rn_cached[1] + r3*rn_cached[2])/r;
	rd[0] = r1/r;
	rd[1] = r2/r;
	rd[2] = r3/r;
	

	zwi = thrust::complex<float>(0, fr);
	
	zc0 = 1.f/(4.f*(pi)*(zge));
	zc1 = ((zcp)/(zcs))*((zcp)/(zcs));
	zc2 = ((zcs)/(zcp))*((zcs)/(zcp));
	zkp = -zwi/(zcp);
	zks = -zwi/(zcs);
	zzp = zkp*r;
	zzs = zks*r;
	zezp= exp(zzp);
	zezs= exp(zzs);
	zp2 = zzp*zzp;
	zs2 = zzs*zzs;

	zfhi    = (1.f + 1.f/zs2 - 1.f/zzs)*zezs/r - zc2*(1.f/zp2 - 1.f/zzp)*zezp/r;
	zcappa  = (1.f + 3.f/zs2 - 3.f/zzs)*zezs/r - zc2*(1.f + 3.f/zp2 - 3.f/zzp)*zezp/r;
	zfhidr  = (-2.f+ zzs + 3.f/zzs - 3.f/zs2)*zezs/(r*r) - zc2*(-1.f + 3.f/zzp - 3.f/zp2)*zezp/(r*r);
	zcappadr= (zzs - 4.f + 9.f/zzs - 9.f/zs2)*zezs/(r*r) - zc2*(zzp - 4.f + 9.f/zzp - 9.f/zp2)*zezp/(r*r);

	zaa = zfhidr-zcappa/r;
	zbb = 4.f*zcappa/r - 2.f*zcappadr;
	zcc = (zc1-2.f)*(zaa + 0.5f*zbb-3.0f*zcappa/r)-2.0f*zcappa/r;

	p12 = p1*p2*det;
	
    if (ii != jj)
    {
        for (j = 0; j < 3; ++j)
        {	for (i = 0; i < 3; ++i)
            {
                zgi = (zc0*(zfhi*delta[j][i] - zcappa*rd[j]*rd[i]));
                

                zhi = (1.0f/(4.0f*pi))*((zaa*(drn*delta[j][i] + 
                                    rd[j]*rn_cached[i])) + rd[i]*rd[j]*drn*zbb + 
                            rd[i]*rn_cached[j]*zcc);
            
                if (ii == jj && !interec)
                {
                    zgi = zgi - (c1/r)*(c2*delta[j][i] + rd[i]*rd[j]);
                    zhi = zhi - (c3/(r*r))*(drn*(c4*delta[j][i] + 3.0f*rd[i]*rd[j]) + c4*(rd[j]*rn_cached[i] - rd[i]*rn_cached[j]));
                }
                
                zgi = zgi*p12;
                zhi = zhi*p12;

                zgelem(j, i, jg*npg + ig) = zgi;
                zhelem(j, i, jg*npg + ig) = zhi;
            }
        }
    }
	__syncthreads();
	
	if (jg < 3 && ig < 3)
	{
		int index = 3*blockIdx.y + (3*nbe)*3*blockIdx.x + ig + (3*nbe)*jg;

		zg[index] = thrust::reduce(thrust::seq, &zgelem(jg, ig, 0), &zgelem(jg, ig, npg*npg));
		zh[index] = thrust::reduce(thrust::seq, &zhelem(jg, ig, 0), &zhelem(jg, ig, npg*npg));
	}
}


void cuda_ghmatecd_(int* nbe,
                    int* npg,
                    int* n,
                    int* np,
                    thrust::complex<float>* zge,
                    thrust::complex<float>* zcs,
                    thrust::complex<float>* zcp,
                    float* c1,
                    float* c2,
                    float* c3,
                    float* c4,
                    float* fr,
                    float* zhest_,
                    float* zgest_,
                    thrust::complex<float>* zgp_,
                    thrust::complex<float>* zhp_,
                    int* status
                   )
{
	
	size_t column_size = 2*(3*(*nbe))*sizeof(thrust::complex<float>);
	
	int shared_mem_size = 2*3*3*(*npg)*(*npg)*sizeof(thrust::complex<float>);
	hipError_t error;
	
	thrust::complex<float>* device_zh;
	thrust::complex<float>* device_zg;

	int* device_return_status;
	int return_status;

	/*Cast os parâmetros de volta para o tipo original*/
	float (*zhest)[3*(*nbe)] = (float (*)[3*(*nbe)]) zhest_;
	float (*zgest)[3*(*nbe)] = (float (*)[3*(*nbe)]) zgest_;
	thrust::complex<float> (*zgp)[3*(*nbe)] = (thrust::complex<float> (*)[3*(*nbe)]) zgp_;
	thrust::complex<float> (*zhp)[3*(*nbe)] = (thrust::complex<float> (*)[3*(*nbe)]) zhp_;

	int i, ii, iterations, width;
	dim3 threadsPerBlock(*npg,*npg);

	error = hipMalloc(&device_return_status, sizeof(int));
	cuda_assert(error);

	width = largest_possible_width(column_size, *n, &iterations);

	error = hipMalloc(&device_zh, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<float>));
	cuda_assert(error);

	error = hipMalloc(&device_zg, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<float>));
	cuda_assert(error);

	for (i = 0; i < iterations; ++i)
	{
		int starting_column = width*i;
		if (starting_column + width > *n)
			width = *n - starting_column;
		dim3 numBlocks(width, *nbe);


		error = hipMemset(device_return_status, 0, sizeof(int));
		cuda_assert(error);

		error = hipMemset(device_zh, 0, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<float>));
		cuda_assert(error);

		error = hipMemset(device_zg, 0, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<float>));
		cuda_assert(error);

		hipDeviceSynchronize();
		ghmatecd_kernel<<<numBlocks, threadsPerBlock, shared_mem_size>>>(
							device_cone,
							device_cx,
							device_cy,
							device_cz,
							device_cxm,
							device_cym,
							device_czm,
							device_zh,
							device_zg,
							(float (*)[3]) device_etas,
							*zge,
							*zcs,
							*zcp,
							*fr,
							device_gi,
							device_ome,
							*c1,
							*c2,
							*c3,
							*c4,
							*npg,
							*n,
							*nbe,
							0,
							*n,
							starting_column,
							device_return_status
							);
		hipDeviceSynchronize();

		error = hipMemcpy(&return_status, device_return_status, sizeof(int), hipMemcpyDeviceToHost);
		cuda_assert(error);

		if (return_status != 0)
		{
			fputs("Matriz Singular\n", stderr);
		}

		error = hipMemcpy(&zhp[3*starting_column], device_zh, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
		cuda_assert(error);
		error = hipMemcpy(&zgp[3*starting_column], device_zg, (3*(*nbe))*(3*(width))*sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);
		cuda_assert(error);

	}

	error = hipFree(device_zh);
	cuda_assert(error);
	error = hipFree(device_zg);
	cuda_assert(error);
	*status = return_status;
    error = hipFree(device_return_status);
    cuda_assert(error);
}
}
