#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "cuda_complex.hpp"

#define restrict __restrict__


extern "C"{

__device__ double myAtomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
		 unsigned long long int old = *address_as_ull, assumed;
		 do {
		     assumed = old;
			 old = atomicCAS(address_as_ull, assumed,
			 __double_as_longlong(val + __longlong_as_double(assumed)));
	     } while (assumed != old);
	    return __longlong_as_double(old);
}

void cuda_assert(hipError_t error)
{
	if (error != hipSuccess)
	{
		fputs(hipGetErrorString(hipGetLastError()), stderr);
		exit(1);
	}
}


void sing_de_(complex<double> zhelem[3][3],
              complex<double> zgelem[3][3],
              double co[][4],
              double* cxm,
              double* cym,
              double* czm,
              double eta[],
              complex<double>* zge,
              complex<double>* zcs,
              complex<double>* zcp,
              double* c1,
              double* c2,
              double* c3,
              double* c4,
              double delta[][3],
              double* pi,
              double* fr,
			  double gi[],
			  double ome[],
              int* npg
              );

void nonsingd_(complex<double> zhelem[3][3],
               complex<double> zgelem[3][3],
               double co[][4],
               double* cxm,
               double* cym,
               double* czm,
               double eta[],
               complex<double>* zge,
               complex<double>* zcs,
               complex<double>* zcp,
               double delta[][3],
               double* pi,
               double* fr,
               int* npg
               );


__global__ void nonsingd_kernel(
						   complex<float> zhelem[3][3],
						   complex<float> zgelem[3][3],
						   double co[3][4],
						   double cxp,
						   double cyp,
						   double czp,
						   double rn[3],
						   complex<double> zge,
						   complex<double> zcs,
						   complex<double> zcp,
						   double delta[3][3],
						   double pi,
						   double fr,
						   double gi[],
						   double ome[],
						   int npg
						   )
{

	const int ig = blockIdx.y;
	const int jg = blockIdx.x;


	double p[4][2], xj[3][2], f[4];
	double g0, g1, g2, p1, p2, p12, sp, sm, rp, rm, temp, det;
    double cxg, cyg, czg; //, cxp, cyp, czp;
    double j1, j2, j3;
    double r1, r2, r3, r, drn, rd[3];
    complex<double> zwi, zc0, zc1, zc2, zkp, zks, zzp, zzs, zezp, zezs, 
                    zp2, zs2, zfhi, zcappa, zfhidr, zcappadr, zaa, zbb, zcc;

	complex<float> zhi, zgi;

	//const double pi  = 3.141592654;
	
	int ii, jj;
/*
	n1 = cone[ne*0 + j];
	n2 = cone[ne*1 + j];
	n3 = cone[ne*2 + j];
	n4 = cone[ne*3 + j];

	co[0][0] = cx[n1 - 1];
	co[1][0] = cy[n1 - 1];
	co[2][0] = cz[n1 - 1];
	co[0][1] = cx[n2 - 1];
	co[1][1] = cy[n2 - 1];
	co[2][1] = cz[n2 - 1];
	co[0][2] = cx[n3 - 1];
	co[1][2] = cy[n3 - 1];
	co[2][2] = cz[n3 - 1];
	co[0][3] = cx[n4 - 1];
	co[1][3] = cy[n4 - 1];
	co[2][3] = cz[n4 - 1];

	cxp = cxm[i];
	cyp = cym[i];
	czp = czm[i];
*/
	zhelem[threadIdx.x][threadIdx.y] = 0;
	zgelem[threadIdx.x][threadIdx.y] = 0;
    
	__syncthreads();

	g2 = gi[jg];
	p2 = ome[jg];
	sp = 1 + g2;
	sm = 1 - g2;
	p[0][0] = -0.25*sm;
	p[1][0] =  0.25*sm;
	p[2][0] =  0.25*sp;
	p[3][0] = -0.25*sp;

	g1 = gi[ig];
	p1 = ome[ig];
	rp = 1 + g1;
	rm = 1 - g1;
	f[0] = 0.25*rm*sm;
	f[1] = 0.25*rp*sm;
	f[2] = 0.25*rp*sp;
	f[3] = 0.25*rm*sp;
	p[0][1] = -0.25*rm;
	p[1][1] = -0.25*rp;
	p[2][1] = 0.25*rp;
	p[3][1] = 0.25*rm;

	for (ii = 0; ii < 2; ++ii)
	{
		for (jj = 0; jj < 3; ++jj)
		{
			xj[jj][ii] = p[0][ii]*co[jj][0] + p[1][ii]*co[jj][1]+ p[2][ii]*co[jj][2] + p[3][ii]*co[jj][3];
		}
	}

    j1 = xj[1][0]*xj[2][1]-xj[1][1]*xj[2][0];
	j2 = xj[0][1]*xj[2][0]-xj[0][0]*xj[2][1];
	j3 = xj[0][0]*xj[1][1]-xj[0][1]*xj[1][0];

	det = sqrt(j1*j1 + j2*j2 + j3*j3);

/*
	if (det < 1e-5)
	{
		//algo deveria acontecer
	}
*/

	cxg = 0;
	cyg = 0;
	czg = 0;
	for (ii = 0; ii < 4; ++ii)
	{
		cxg = cxg + co[0][ii]*f[ii];
		cyg = cyg + co[1][ii]*f[ii];
		czg = czg + co[2][ii]*f[ii];
	}


	r1    = cxg - cxp;
	r2    = cyg - cyp;
	r3    = czg - czp;
	r     = sqrt(r1*r1 + r2*r2 + r3*r3);
	drn   = (r1*rn[0] + r2*rn[1] + r3*rn[2])/r;
	rd[0] = r1/r;
	rd[1] = r2/r;
	rd[2] = r3/r;

	zwi = complex<double>(0, fr);
	
	zc0 = 1.0/(4*(pi)*(zge));
	zc1 = ((zcp)/(zcs))*((zcp)/(zcs));
	zc2 = ((zcs)/(zcp))*((zcs)/(zcp));
	zkp = -zwi/(zcp);
	zks = -zwi/(zcs);
	zzp = zkp*r;
	zzs = zks*r;
	zezp= exp(zzp);
	zezs= exp(zzs);
	zp2 = zzp*zzp;
	zs2 = zzs*zzs;

	zfhi    = (1. + 1./zs2 - 1./zzs)*zezs/r - zc2*(1./zp2 - 1./zzp)*zezp/r;
	zcappa  = (1. + 3./zs2 - 3./zzs)*zezs/r - zc2*(1. + 3./zp2 - 3./zzp)*zezp/r;
	zfhidr  = (-2.+ zzs + 3./zzs - 3./zs2)*zezs/(r*r) - zc2*(-1. + 3./zzp - 3./zp2)*zezp/(r*r);
	zcappadr= (zzs - 4. + 9./zzs - 9./zs2)*zezs/(r*r) - zc2*(zzp - 4. + 9./zzp - 9./zp2)*zezp/(r*r);

	zaa = zfhidr-zcappa/r;
	zbb = 4.*zcappa/r - 2.*zcappadr;
	zcc = (zc1-2.)*(zaa + 0.5*zbb-3.0*zcappa/r)-2.0*zcappa/r;

	p12 = p1*p2*det;

	zgi = (zc0*(zfhi*delta[threadIdx.x][threadIdx.y] - zcappa*rd[threadIdx.x]*rd[threadIdx.y])*zzp)*p12;
	zhi = (1.0/(4.0*pi))*((zaa*(drn*delta[threadIdx.x][threadIdx.y] + 
						rd[threadIdx.x]*rn[threadIdx.y])) + rd[threadIdx.y]*rd[threadIdx.x]*drn*zbb + 
				rd[threadIdx.y]*rn[threadIdx.x]*zcc)*p12;

	atomicAdd((float*) &zgelem[threadIdx.x][threadIdx.y]                , real(zgi));
	atomicAdd((float*) &zgelem[threadIdx.x][threadIdx.y] + sizeof(float), imag(zgi));

	atomicAdd((float*) &zhelem[threadIdx.x][threadIdx.x]                , real(zhi));
	atomicAdd((float*) &zhelem[threadIdx.x][threadIdx.x] + sizeof(float), imag(zhi));
}


__global__ void ghmatecd_kernel(int ne,
		                   int nbe,
						   int nx,
						   int npg,
						   int ncox,
						   int n,
						   complex<double>* zh,
						   complex<double>* zg,
						   complex<double> zge,
						   complex<double> zcs,
						   complex<double> zcp,
						   double* zhest_,
						   double* zgest_,
						   double cx[],
						   double cy[],
						   double cz[],
						   double* cxm,
						   double* cym,
						   double* czm,
						   double gi[],
						   double ome[],
						   double delta[3][3],
						   double rn[][3],
						   double fr,
						   int* cone
						   )
{
	const int i = blockIdx.y;
	const int j = blockIdx.x;
	const int index = 3*blockIdx.y + 3*blockIdx.x*nx + threadIdx.x*nx + threadIdx.y;
	
	__shared__ double p[4][2], xj[3][2], f[4];
	double g0, g1, g2, p1, p2, p12, sp, sm, rp, rm, temp, det;
    double cxg, cyg, czg, cxp, cyp, czp;
    double j1, j2, j3;
	double co[3][4];
    double r1, r2, r3, r, drn, rd[3];
    complex<double> zwi, zc0, zc1, zc2, zkp, zks, zzp, zzs, zezp, zezs, 
                    zp2, zs2, zfhi, zcappa, zfhidr, zcappadr, zaa, zbb, zcc;

	
	const double pi  = 3.141592654;
	
	__shared__ int n1;
	__shared__ int n2;
	__shared__ int n3;
	__shared__ int n4;

	n1 = cone[ne*0 + j];
	n2 = cone[ne*1 + j];
	n3 = cone[ne*2 + j];
	n4 = cone[ne*3 + j];

	int ig, jg, ii, jj;

	co[0][0] = cx[n1 - 1];
	co[1][0] = cy[n1 - 1];
	co[2][0] = cz[n1 - 1];
	co[0][1] = cx[n2 - 1];
	co[1][1] = cy[n2 - 1];
	co[2][1] = cz[n2 - 1];
	co[0][2] = cx[n3 - 1];
	co[1][2] = cy[n3 - 1];
	co[2][2] = cz[n3 - 1];
	co[0][3] = cx[n4 - 1];
	co[1][3] = cy[n4 - 1];
	co[2][3] = cz[n4 - 1];

	cxp = cxm[i];
	cyp = cym[i];
	czp = czm[i];

	zh[index] = 0;
	zg[index] = 0;
    
	for (jg = 0; jg < npg; ++jg)
	{
	
		g2 = gi[jg];
		p2 = ome[jg];
		sp = 1 + g2;
		sm = 1 - g2;
		p[0][0] = -0.25*sm;
		p[1][0] =  0.25*sm;
		p[2][0] =  0.25*sp;
		p[3][0] = -0.25*sp;

		for (ig = 0; ig < npg; ++ig)
		{
			g1 = gi[ig];
			p1 = ome[ig];
			rp = 1 + g1;
			rm = 1 - g1;
			f[0] = 0.25*rm*sm;
			f[1] = 0.25*rp*sm;
			f[2] = 0.25*rp*sp;
			f[3] = 0.25*rm*sp;
			p[0][1] = -0.25*rm;
			p[1][1] = -0.25*rp;
			p[2][1] = 0.25*rp;
			p[3][1] = 0.25*rm;

			__syncthreads();

			xj[threadIdx.x][threadIdx.y] = p[0][threadIdx.y]*co[threadIdx.x][0] + p[1][threadIdx.y]*co[threadIdx.x][1]+ p[2][threadIdx.y]*co[threadIdx.x][2] + p[3][threadIdx.y]*co[threadIdx.x][3];

			__syncthreads();
			/*
            for (ii = 0; ii < 2; ++ii)
            {
                for (jj = 0; jj < 3; ++jj)
                {
                    xj[jj][ii] = p[0][ii]*co[jj][0] + p[1][ii]*co[jj][1]+ p[2][ii]*co[jj][2] + p[3][ii]*co[jj][3];
                }
            }
*/
            j1 = xj[1][0]*xj[2][1]-xj[1][1]*xj[2][0];
            j2 = xj[0][1]*xj[2][0]-xj[0][0]*xj[2][1];
            j3 = xj[0][0]*xj[1][1]-xj[0][1]*xj[1][0];

            det = sqrt(j1*j1 + j2*j2 + j3*j3);

/*
            if (det < 1e-5)
            {
                //algo deveria acontecer
            }
*/

			cxg = 0;
			cyg = 0;
			czg = 0;
			for (ii = 0; ii < 4; ++ii)
			{
				cxg = cxg + co[0][ii]*f[ii];
				cyg = cyg + co[1][ii]*f[ii];
				czg = czg + co[2][ii]*f[ii];
			}


			r1    = cxg - cxp;
			r2    = cyg - cyp;
			r3    = czg - czp;
			r     = sqrt(r1*r1 + r2*r2 + r3*r3);
			drn   = (r1*rn[j][0] + r2*rn[j][1] + r3*rn[j][2])/r;
			rd[0] = r1/r;
			rd[1] = r2/r;
			rd[2] = r3/r;

			zwi = complex<double>(0, fr);
			
			zc0 = 1.0/(4*(pi)*(zge));
			zc1 = ((zcp)/(zcs))*((zcp)/(zcs));
			zc2 = ((zcs)/(zcp))*((zcs)/(zcp));
			zkp = -zwi/(zcp);
			zks = -zwi/(zcs);
			zzp = zkp*r;
			zzs = zks*r;
			zezp= exp(zzp);
			zezs= exp(zzs);
			zp2 = zzp*zzp;
			zs2 = zzs*zzs;

			zfhi    = (1. + 1./zs2 - 1./zzs)*zezs/r - zc2*(1./zp2 - 1./zzp)*zezp/r;
			zcappa  = (1. + 3./zs2 - 3./zzs)*zezs/r - zc2*(1. + 3./zp2 - 3./zzp)*zezp/r;
			zfhidr  = (-2.+ zzs + 3./zzs - 3./zs2)*zezs/(r*r) - zc2*(-1. + 3./zzp - 3./zp2)*zezp/(r*r);
			zcappadr= (zzs - 4. + 9./zzs - 9./zs2)*zezs/(r*r) - zc2*(zzp - 4. + 9./zzp - 9./zp2)*zezp/(r*r);

			zaa = zfhidr-zcappa/r;
			zbb = 4.*zcappa/r - 2.*zcappadr;
			zcc = (zc1-2.)*(zaa + 0.5*zbb-3.0*zcappa/r)-2.0*zcappa/r;

			p12 = p1*p2*det;

			zg[index] += (zc0*(zfhi*delta[threadIdx.x][threadIdx.y] - zcappa*rd[threadIdx.x]*rd[threadIdx.y]))*p12;
		    zh[index] += ((1.0/(4.0*pi))*((zaa*(drn*delta[threadIdx.x][threadIdx.y] + rd[threadIdx.x]*rn[blockIdx.x][threadIdx.y])) + rd[threadIdx.y]*rd[threadIdx.x]*drn*zbb + rd[threadIdx.y]*rn[blockIdx.x][threadIdx.x]*zcc))*p12;
		}
	}
}

void cuda_ghmatecd_(int* ne,
                    int* nbe,
                    int* nx,
                    int* npg,
                    int* ncox,
					int* n,
					int* cone_,
					double cx[],
                    double cy[],
                    double cz[],
                    double cxm[],
                    double cym[],
                    double czm[],
                    double etas[][3],
                    complex<double>* zge,
                    complex<double>* zcs,
                    complex<double>* zcp,
                    double* c1,
                    double* c2,
                    double* c3,
                    double* c4,
                    double delta[3][3],
                    double* fr,
                    double* zhest_,
					double* zgest_,
					complex<double>* zgp_,
                    complex<double>* zhp_,
					double ome[],
					double* gi,
					int* status
                   )
{
	dim3 threadsPerBlock(3,3);
	dim3 numBlocks(*n, *nbe);
	hipError_t error;

    double co[3][4];
    int n1, n2, n3, n4;
    int i, j, ii, jj;
    complex<double> zgelem[3][3];
	complex<double> zhelem[3][3];
	complex<float> zgelemf[3][3];
	complex<float> zhelemf[3][3];

	complex<float>* device_zhelem;
	complex<float>* device_zgelem;
	double* device_gi;
	double* device_ome;
	double* device_etas;
	double* device_delta;
	double* device_co;


	/*Cast os parâmetros de volta para o tipo original*/
	int (*cone)[*ne]           = (int (*)[*ne]) cone_;
	double (*zhest)[*nx]       = (double (*)[*nx]) zhest_;
	double (*zgest)[*nx]       = (double (*)[*nx]) zgest_;
	complex<double> (*zgp)[*nx] = (complex<double> (*)[*nx]) zgp_;
	complex<double> (*zhp)[*nx] = (complex<double> (*)[*nx]) zhp_;

	double pi  = 3.141592654;

	error = hipMalloc(&device_zhelem, 3*3*sizeof(complex<float>));
	cuda_assert(error);

	error = hipMalloc(&device_zgelem, 3*3*sizeof(complex<float>));
	cuda_assert(error);

	error = hipMalloc(&device_gi, (*npg)*sizeof(double));
	cuda_assert(error);

	error = hipMalloc(&device_ome, (*npg)*sizeof(double));
	cuda_assert(error);
	
	error = hipMalloc(&device_etas, (*nx)*3*sizeof(double));
	cuda_assert(error);
	
	error = hipMalloc(&device_delta, 3*3*sizeof(double));
	cuda_assert(error);

	error = hipMalloc(&device_co,  3*4*sizeof(double));
	cuda_assert(error);
	
	error = hipMemcpy(device_gi, gi, (*npg)*sizeof(double), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_ome, ome, (*npg)*sizeof(double), hipMemcpyHostToDevice);
	cuda_assert(error);

	error = hipMemcpy(device_etas, etas, (*n)*3*sizeof(double), hipMemcpyHostToDevice);
	cuda_assert(error);
	

	for (j = 0; j < *n; ++j)
    {
		n1 = cone[0][j];
        n2 = cone[1][j];
        n3 = cone[2][j];
        n4 = cone[3][j];

        co[0][0] = cx[n1 - 1];
        co[1][0] = cy[n1 - 1];
        co[2][0] = cz[n1 - 1];
        co[0][1] = cx[n2 - 1];
        co[1][1] = cy[n2 - 1];
        co[2][1] = cz[n2 - 1];
        co[0][2] = cx[n3 - 1];
        co[1][2] = cy[n3 - 1];
        co[2][2] = cz[n3 - 1];
        co[0][3] = cx[n4 - 1];
        co[1][3] = cy[n4 - 1];
        co[2][3] = cz[n4 - 1];

		error = hipMemcpy(device_co, (double*) co,  3*4*sizeof(double), hipMemcpyHostToDevice);
		cuda_assert(error);
        
		jj = 3*j;
        for (i = 0; i < *nbe; ++i)
        {
            ii = 3*i;

			printf("i = %d, j = %d\n", i, j);
            if (i == j)
            {
                sing_de_(zhelem, 
                         zgelem, 
                         co, 
                         &cxm[i], 
                         &cym[i], 
                         &czm[i], 
                         etas[j],
                         zge,
                         zcs,
                         zcp,
                         c1,
                         c2,
                         c3,
                         c4,
                         delta,
                         &pi,
                         fr,
						 gi,
						 ome,
                         npg
                        );
                
				for (int jjj = 0; jjj < 3; ++jjj)
				{   for (int iii = 0; iii < 3; ++iii)
                    {
                        zgp[jjj+jj][iii+ii] = zgelem[jjj][iii] + zgest[jjj+jj][iii+ii];
                        zhp[jjj+jj][iii+ii] = zhelem[jjj][iii] + zhest[jjj+jj][iii+ii];
                    }
				}
            }
            else
            {
				nonsingd_kernel<<<numBlocks, threadsPerBlock>>>(
						(complex<float> (*)[3]) device_zhelem,
						(complex<float> (*)[3]) device_zgelem,
						(double (*)[4])          device_co,
						cxm[i],
						cym[i],
						czm[i],
						&device_etas[j],
						*zge,
						*zcs,
						*zcp,
						(double (*)[3])device_delta,
						pi,
						*fr,
						device_gi,
						device_ome,
						*npg
						);
				
				/*
                nonsingd(zhelem, 
                          zgelem, 
                          co, 
                          &cxm[i], 
                          &cym[i], 
                          &czm[i],
                          etas[j],
                          zge,
                          zcs,
                          zcp,
                          delta,
                          pi,
                          fr,
                          npg
                         );
			*/
	/*	
				error = hipMemcpy(zgelemf, device_zgelem, 3*3*sizeof(complex<float>), hipMemcpyDeviceToHost);
				cuda_assert(error);
				error = hipMemcpy(zhelemf, device_zhelem, 3*3*sizeof(complex<float>), hipMemcpyDeviceToHost);
				cuda_assert(error);
*/
				for (int jjj = 0; jjj < 3; ++jjj)
				{   for (int iii = 0; iii < 3; ++iii)
                    {
                        zgp[jjj+jj][iii+ii] = zgelemf[jjj][iii];
                        zhp[jjj+jj][iii+ii] = zhelemf[jjj][iii];
                    }
				}
			}
        }
    }
	printf("ACABOU\n");
}
}
