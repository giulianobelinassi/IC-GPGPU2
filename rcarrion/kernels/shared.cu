#include "shared.h"
#include <cstdio>

extern "C"{

FREAL* device_cx;
FREAL* device_cy;
FREAL* device_cz;
FREAL* device_cxm;
FREAL* device_cym;
FREAL* device_czm;
FREAL* device_etas;
FREAL* device_gi;
FREAL* device_ome;
int*   device_cone;

/*Disponivel apos a execucao de Ghmatece*/
FREAL* device_hestdiag;
FREAL* device_gestdiag;
/**/
/*Disponivel apos a execucao de Ghmatecd, caso fastsing esteja ligado*/
thrust::complex<FREAL>* device_zh;
thrust::complex<FREAL>* device_zg;

void cuda_assert(hipError_t error)
{
    if (error != hipSuccess)
    {   
        fputs(hipGetErrorString(hipGetLastError()), stderr);
        putc('\n', stderr);
        exit(1);
    }   
}

/*https://stackoverflow.com/questions/13041399/equivalent-of-cudageterrorstring-for-cublas*/
void cublas_assert(hipblasStatus_t error)
{
	switch (error)
    {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            fputs("HIPBLAS_STATUS_NOT_INITIALIZED\n", stderr);
			exit(1);

        case HIPBLAS_STATUS_ALLOC_FAILED:
            fputs("HIPBLAS_STATUS_ALLOC_FAILED\n", stderr);
			exit(1);
        
		case HIPBLAS_STATUS_INVALID_VALUE:
            fputs("HIPBLAS_STATUS_INVALID_VALUE\n", stderr);
			exit(1);

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            fputs("HIPBLAS_STATUS_ARCH_MISMATCH\n", stderr);
			exit(1);

        case HIPBLAS_STATUS_MAPPING_ERROR:
            fputs("HIPBLAS_STATUS_MAPPING_ERROR\n", stderr);
			exit(1);

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            fputs("HIPBLAS_STATUS_EXECUTION_FAILED\n", stderr);
			exit(1);

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            fputs("HIPBLAS_STATUS_INTERNAL_ERROR\n", stderr);
			exit(1);
		
		case HIPBLAS_STATUS_SUCCESS:
			return;
    }
	fputs("CUBLAS: Erro desconhecido.\n", stderr);
	exit(1);

}

void send_shared_data_to_gpu_(
        FREAL cx[],
        FREAL cy[],
        FREAL cz[],
        FREAL cxm[],
        FREAL cym[],
        FREAL czm[],
        FREAL etas[],
        FREAL gi[],
        FREAL ome[],
        int cone[],
        int* np, 
        int* npg,
        int* n,
        int* nbe 
        )
{
    hipError_t error;

    /*Aloque memória para os vetores na GPU*/
    error = hipMalloc(&device_cone, 4*(*n)*sizeof(int));
    cuda_assert(error);

    error = hipMalloc(&device_cx, (*np)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_cy, (*np)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_cz, (*np)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_cxm, (*n)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_cym, (*n)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_czm, (*n)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_gi, (*npg)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_ome, (*npg)*sizeof(FREAL));
    cuda_assert(error);

    error = hipMalloc(&device_etas, (*n)*3*sizeof(FREAL));
    cuda_assert(error);

    /*mova os dados para lá*/
    error = hipMemcpy(device_cone, cone, 4*(*n)*sizeof(int), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_cx, cx, (*np)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_cy, cy, (*np)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_cz, cz, (*np)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_cxm, cxm, (*n)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_cym, cym, (*n)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_czm, czm, (*n)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_gi, gi, (*npg)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_ome, ome, (*npg)*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

    error = hipMemcpy(device_etas, etas, (*n)*3*sizeof(FREAL), hipMemcpyHostToDevice);
    cuda_assert(error);

}

void deallocate_shared_gpu_data_()
{
    hipError_t error;

    error = hipFree(device_cone);
    cuda_assert(error);
    error = hipFree(device_gi);
    cuda_assert(error);
    error = hipFree(device_ome);
    cuda_assert(error);
    error = hipFree(device_etas);
    cuda_assert(error);
    error = hipFree(device_cx);
    cuda_assert(error);
    error = hipFree(device_cz);
    cuda_assert(error);
    error = hipFree(device_cxm);
    cuda_assert(error);
    error = hipFree(device_cym);
    cuda_assert(error);
    error = hipFree(device_czm);
    cuda_assert(error);
    error = hipFree(device_hestdiag);
    cuda_assert(error);
    error = hipFree(device_gestdiag);
    cuda_assert(error);
}

int largest_possible_width(size_t sizeof_column_mem, int columns, int* iterations)
{
	size_t available_mem;
	size_t total_mem;
	int possible_width;

	cuda_assert(hipMemGetInfo(&available_mem, &total_mem));
	available_mem = available_mem;
//	available_mem = 8*1024*1024; //Simulate a GPU with 8Mb of video memory

	if ((3*columns)*sizeof_column_mem < available_mem)
	{	*iterations = 1;
		return columns;
	}
	possible_width = available_mem/(3*sizeof_column_mem);

	*iterations = (columns + possible_width - 1)/possible_width;

	return possible_width;
}

}
